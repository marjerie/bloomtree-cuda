#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <inttypes.h>

#define FORCE_INLINE

__device__ int Parent(int node);
__device__ int LeftChild(int node);
__device__ int RightChild(int node);
__device__ int Sibling(int node);

__device__ static inline FORCE_INLINE uint64_t rotl64 ( uint64_t x, int8_t r )
{
  return (x << r) | (x >> (64 - r));
}

#define ROTL64(x,y)	rotl64(x,y)
#define BIG_CONSTANT(x) (x##LLU)

#define getblock(p, i) (p[i])

__device__ static inline FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

__device__ void MurmurHash3_x64_128 ( const void * key, const int len,
                           const uint32_t seed, void * out1 , void * out2)
{
  const uint8_t * data = (const uint8_t*)key;
  const int nblocks = len / 16;
  int i;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //----------
  // body

  const uint64_t * blocks = (const uint64_t *)(data);

  for(i = 0; i < nblocks; i++)
  {
    uint64_t k1 = getblock(blocks,i*2+0);
    uint64_t k2 = getblock(blocks,i*2+1);

    k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;

    h1 = ROTL64(h1,27); h1 += h2; h1 = h1*5+0x52dce729;

    k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    h2 = ROTL64(h2,31); h2 += h1; h2 = h2*5+0x38495ab5;
  }

  //----------
  // tail

  const uint8_t * tail = (const uint8_t*)(data + nblocks*16);

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15)
  {
  case 15: k2 ^= (uint64_t)(tail[14]) << 48;
  case 14: k2 ^= (uint64_t)(tail[13]) << 40;
  case 13: k2 ^= (uint64_t)(tail[12]) << 32;
  case 12: k2 ^= (uint64_t)(tail[11]) << 24;
  case 11: k2 ^= (uint64_t)(tail[10]) << 16;
  case 10: k2 ^= (uint64_t)(tail[ 9]) << 8;
  case  9: k2 ^= (uint64_t)(tail[ 8]) << 0;
           k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

  case  8: k1 ^= (uint64_t)(tail[ 7]) << 56;
  case  7: k1 ^= (uint64_t)(tail[ 6]) << 48;
  case  6: k1 ^= (uint64_t)(tail[ 5]) << 40;
  case  5: k1 ^= (uint64_t)(tail[ 4]) << 32;
  case  4: k1 ^= (uint64_t)(tail[ 3]) << 24;
  case  3: k1 ^= (uint64_t)(tail[ 2]) << 16;
  case  2: k1 ^= (uint64_t)(tail[ 1]) << 8;
  case  1: k1 ^= (uint64_t)(tail[ 0]) << 0;
           k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)out1)[0] = h1;
  ((uint64_t*)out2)[0] = h2;

}

__global__ void init_mask(bool *mask, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;
	if (tid < 2*n*(n-1))
		mask[tid] = 0;
}
__global__ void adjust_level(int *u, int *v, bool *mask, int n, int e, long int ful_vertices)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid < e){
		u[tid] = u[tid] + n - 1;
		v[tid] = v[tid] + n - 1;
		__syncthreads();
		if (!((u[tid] < ful_vertices && v[tid] < ful_vertices) || (u[tid] >= ful_vertices && v[tid] >= ful_vertices))) {
			if (u[tid] > v[tid]){
				int src = u[tid];
				int dest = v[tid];
	 			int cur = Parent(u[tid]);
				mask[(cur*n+src-n+1) << 1] = 1;
				if (src == LeftChild(cur))
					mask[(cur*n+dest-n+1) << 1] = 1;
				else
					mask[((cur*n+dest-n+1) << 1) + 1] = 1;
				u[tid] = cur;
			}	
		 	else{
				int src = v[tid];
				int dest = u[tid];
	 			int cur = Parent(v[tid]);
				mask[(cur*n+src-n+1) << 1] = 1;
				if (src == LeftChild(cur))
					mask[(cur*n+dest-n+1) << 1] = 1;
				else
					mask[((cur*n+dest-n+1) << 1) + 1] = 1;
				v[tid] = cur;
			}
		}
	}
}

__global__ void calculate_lca(int *u, int *v, int *lca, int e)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	int val1 = 0;
	int val2 = 0;	

	if (tid < e){
		int i = 1;
		do{
			float pow_val = 1 << i;
			val1 = floor((u[tid]+1)/pow_val);
			val2 = floor((v[tid]+1)/pow_val);
			i++;
		} while(val1 != val2);
		lca[tid] = val1 - 1;
	}
}

__global__ void traversal(int *u, int *lca, int *source, int *destination, bool *mask, int n, int e)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;		

	if (tid < e){
		int prev = u[tid];
		int cur = Parent(u[tid]);
		int src = source[tid];
		int dest = destination[tid];
		while (cur != lca[tid]){
			mask[(cur*n+src) << 1] = 1;
			if (prev == LeftChild(cur))
				mask[(cur*n+dest) << 1] = 1;
			else
				mask[((cur*n+dest) << 1) + 1] = 1;
			prev = cur;
			cur = Parent(cur);
		}
		mask[((cur*n+src) << 1) + 1] = 1;
		mask[((cur*n+dest) << 1) + 1] = 1;
	}	
}

__global__ void to_str(char *str, bool *mask, int n, int count)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid<n){
		int num=tid;
		int val=0;
		int i =0;
		int len=0;
		do{
			len++;
			num/=10;
		}while(num !=0);
		num = tid;
		do{
			val=num%10 + 48;
			num/=10;
			char *op = (char *) (str+tid*sizeof(char)*(count+1)+len-i-1);
			*op = val;
			i++;
		}while(num !=0);
		char *op = (char *) (str+tid*sizeof(char)*(count+1)+len);
		*op = 48+'\0';
	}
}

__global__ void get_hash(bool *mask, uint64_t *hash_value, char *array, int n, int count)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 2*n*(n-1)){
		if (mask[tid] == 1){
			int count1 = 0; 	
			int num = tid;
			do{
				count1++;
				num /= 10;
			} while(num != 0);
			uint64_t len1 = (uint64_t) count1;
			size_t len = (size_t) len1;
			char *a;
			a = (char*) (array+tid*sizeof(char)*(count+1));
			printf("value of a : %c and tid is %d\n",*a,tid);
			MurmurHash3_x64_128(a, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));
		}
	}
}

__global__ void print_hash(uint64_t *hash_value, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 4*n*(n-1)){
		//printf("%u\n",*(hash_value+tid*sizeof(uint8_t)));
		printf("%" PRIu64 " %d\n",*(hash_value+tid*sizeof(uint64_t)),tid);
	}	
}

__device__ int Parent(int node)
{
	return (((node + 1) >> 1) - 1); 
}

__device__ int LeftChild(int node)
{
	return (((node + 1) << 1) - 1); 
}

__device__ int RightChild(int node)
{
	return ((node + 1) << 1); 
}

__device__ int Sibling(int node)
{
	return (((node + 1) ^ 1) - 1); 
}

int main ()
{
	int num_vertices, num_edges;
	scanf("%d",&num_vertices);
	scanf("%d",&num_edges);

	size_t size = num_edges * sizeof(int);
	int num_vals = 2*num_vertices*(num_vertices-1);

	int *h_u = (int *)malloc(size);
	int *h_v = (int *)malloc(size);
	
	for (int i =0; i<num_edges; i++)
	{
		scanf("%d",&h_u[i]);
		scanf("%d",&h_v[i]);
	}

	int *d_u = NULL, *d_v = NULL, *d_src = NULL, *d_dest = NULL;
        hipMalloc((void **)&d_u, size);
        hipMalloc((void **)&d_v, size);
        hipMalloc((void **)&d_src, size);
        hipMalloc((void **)&d_dest, size);

	hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
	hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
	hipMemcpy(d_src, h_u, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dest, h_v, size, hipMemcpyHostToDevice);

	size_t size_mask = num_vals * sizeof(bool);
	bool *h_mask = (bool *)malloc(size_mask);
	bool *d_mask = NULL;
        hipMalloc((void **)&d_mask, size_mask);

	dim3 tpb(num_vertices,(2*(num_vertices-1)),1);
	dim3 bpg(1,1,1);

	init_mask<<<bpg,tpb>>>(d_mask,num_vertices);

	int tpb1 = num_edges;
        int bpg1 = 1;
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;

	adjust_level<<<bpg1,tpb1>>>(d_u,d_v,d_mask,num_vertices,num_edges,ful_vertices);	

	//int *h_lca = (int *)malloc(size);
	int *d_lca = NULL;
        hipMalloc((void **)&d_lca, size);
	//hipMemcpy(d_lca, h_lca, size, hipMemcpyHostToDevice);

	calculate_lca<<<bpg1,tpb1>>>(d_u,d_v,d_lca,num_edges);

	traversal<<<bpg1,tpb1>>>(d_u,d_lca,d_src,d_dest,d_mask,num_vertices,num_edges);
	traversal<<<bpg1,tpb1>>>(d_v,d_lca,d_dest,d_src,d_mask,num_vertices,num_edges);

	//thrust::device_vector<uint64_t> d_hash_value(4*num_vertices*(num_vertices-1));
	
	hipMemcpy(h_mask, d_mask, size_mask, hipMemcpyDeviceToHost);	

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);
	uint64_t *h_hash_value = (uint64_t *)malloc(size_hash);

	int num = num_vals;
	int count = 0;
	do{
		count++;
		num /= 10;
	} while(num != 0);
	
	char *d_str = NULL;
	size_t size_str = (count+1)*num_vals*sizeof(char);
	hipMalloc((void **)&d_str, size_str);
	char *h_str = (char *)malloc(size_str);	

	to_str<<<1,num_vals>>>(d_str,d_mask,num_vals,count);

	get_hash<<<bpg,tpb>>>(d_mask,d_hash_value,d_str,num_vertices,count);
	
	dim3 tpb2(num_vertices,(4*(num_vertices-1)),1);
	dim3 bpg2(1,1,1);

	print_hash<<<bpg2,tpb2>>>(d_hash_value,num_vertices);
	
	hipDeviceSynchronize();	
	hipMemcpy(h_hash_value, d_hash_value, size_hash, hipMemcpyDeviceToHost);
	hipMemcpy(h_str, d_str, size_str, hipMemcpyDeviceToHost);

	//printf("printing hash vector\n");
	//for (int i=0; i<num_vertices*4*(num_vertices-1); i++){		
	//	printf("%" PRIu64 "\n", h_hash_value[i]);
	//}

	printf("printing string value\n");
	for (int i=0; i<num_vals; i++){	
		for (int j=0; j<count+1; j++)	
			printf("%c ", *(h_str+((count+1)*i+j)*sizeof(char)));
		printf("\n");
	}

	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_src);
	hipFree(d_dest);

	return 0;
}
