#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdint.h>
#include <inttypes.h>

#define FORCE_INLINE

__device__ int Parent(int node);
__device__ int LeftChild(int node);
__device__ int RightChild(int node);
__device__ int Sibling(int node);
__device__ int calculate_lca(int u, int v);
__device__ void traversal(int prev, int lca, int src, int dest, bool *mask, int n);
// void get_neighs(int *no, int src, int next, int cur,bool dir,bool *bit,int *neigh,uint64_t *hash_value,int n,int num,int m,int h);
__device__ bool check_traversal_up(int prev, int lca, int src, int dest, int n, uint64_t *hash_value, bool *bit, int h, int m);
__device__ bool check_traversal_down(int prev, int lca, int src, int dest, int n, uint64_t *hash_value, bool *bit, int h, int m);
__device__ bool CheckBloom(int tid, uint64_t *hash_value, bool *bit, int h, int m);

__device__ static inline FORCE_INLINE uint64_t rotl64 ( uint64_t x, int8_t r )
{
  return (x << r) | (x >> (64 - r));
}

#define ROTL64(x,y)	rotl64(x,y)
#define BIG_CONSTANT(x) (x##LLU)

#define getblock(p, i) (p[i])

__device__ static inline FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

__device__ void MurmurHash3_x64_128 ( int tid, char key[], const int len,
                           const uint32_t seed, void * out1 , void * out2)
{
  //const uint8_t * data = (const uint8_t*)key;
  //printf("tid = %d, %c, %c, %c\n",tid,key[0],key[1],key[2]);

  //if (key[0]== 48) {printf("h "); key[0] = 48+'\0';}

  uint8_t data[16];
  for (int i=0; i<len; i++)
    data[i] = (uint8_t) key[i];
  const int nblocks = len / 16;
  int i;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //----------
  // body

  //const uint64_t * blocks = (const uint64_t *)(data);

  uint64_t blocks[16];
  for (int i=0; i<len; i++)
    blocks[i] = (uint64_t) data[i];

  for(i = 0; i < nblocks; i++)
  {
    uint64_t k1 = blocks[i*2+0];
    uint64_t k2 = blocks[i*2+1];

    k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;

    h1 = ROTL64(h1,27); h1 += h2; h1 = h1*5+0x52dce729;

    k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    h2 = ROTL64(h2,31); h2 += h1; h2 = h2*5+0x38495ab5;
  }

  //----------
  // tail

  //const uint8_t * tail = (const uint8_t*)(data + nblocks*16);

  uint8_t tail[16];
  for (int i=0; i<len-nblocks*16; i++)
    tail[i] = (uint64_t) data[i+nblocks*16];

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15)
  {
  case 15: k2 ^= (uint64_t)(tail[14]) << 48;
  case 14: k2 ^= (uint64_t)(tail[13]) << 40;
  case 13: k2 ^= (uint64_t)(tail[12]) << 32;
  case 12: k2 ^= (uint64_t)(tail[11]) << 24;
  case 11: k2 ^= (uint64_t)(tail[10]) << 16;
  case 10: k2 ^= (uint64_t)(tail[ 9]) << 8;
  case  9: k2 ^= (uint64_t)(tail[ 8]) << 0;
           k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

  case  8: k1 ^= (uint64_t)(tail[ 7]) << 56;
  case  7: k1 ^= (uint64_t)(tail[ 6]) << 48;
  case  6: k1 ^= (uint64_t)(tail[ 5]) << 40;
  case  5: k1 ^= (uint64_t)(tail[ 4]) << 32;
  case  4: k1 ^= (uint64_t)(tail[ 3]) << 24;
  case  3: k1 ^= (uint64_t)(tail[ 2]) << 16;
  case  2: k1 ^= (uint64_t)(tail[ 1]) << 8;
  case  1: k1 ^= (uint64_t)(tail[ 0]) << 0;
           k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)out1)[0] = h1;
  ((uint64_t*)out2)[0] = h2;

}

__device__ inline uint64_t NthHash(uint8_t n, uint64_t hashA, uint64_t hashB, uint64_t filter_size) {
	//printf("%" PRIu64 " and %" PRIu64 " %u \n",hashA,hashB,((hashA + n * hashB) % filter_size));
	//printf("%u\n",n);
	//printf ("nth %u\n",((hashA + n * hashB) % filter_size));
	return ((hashA + n * hashB) % filter_size);
}

__global__ void init_mask(bool *mask, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;
	if (tid < 2*n*(n-1))
		mask[tid] = 0;
}

__global__ void init_bits(bool *bit, int n)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	if (tid < n)
		bit[tid] = 0;
}


__global__ void get_mask(int *u, int *v, bool *mask, int n, int e, long int ful_vertices)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	if (tid < e){
		u[tid] = u[tid] + n - 1;
		v[tid] = v[tid] + n - 1;
		int src = u[tid];
		int dest = v[tid];

		if (!((u[tid] < ful_vertices && v[tid] < ful_vertices) || (u[tid] >= ful_vertices && v[tid] >= ful_vertices))) {
			if (u[tid] > v[tid]){
	 			int cur = Parent(u[tid]);
				mask[(cur*n+u[tid]-n+1) << 1] = 1;
				if (u[tid] == LeftChild(cur))
					mask[(cur*n+v[tid]-n+1) << 1] = 1;
				else
					mask[((cur*n+v[tid]-n+1) << 1) + 1] = 1;
				u[tid] = cur;
			}	
		 	else{
	 			int cur = Parent(v[tid]);
				mask[(cur*n+v[tid]-n+1) << 1] = 1;
				if (v[tid] == LeftChild(cur))
					mask[(cur*n+u[tid]-n+1) << 1] = 1;
				else
					mask[((cur*n+u[tid]-n+1) << 1) + 1] = 1;
				v[tid] = cur;
			}
		}
		
		__syncthreads();

		int lca = calculate_lca(u[tid], v[tid]);
		traversal(u[tid], lca, src, dest, mask, n);
		traversal(v[tid], lca, dest, src, mask, n);
	}
}

__global__ void get_hash(bool *mask, uint64_t *hash_value, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 2*n*(n-1)){
		if (mask[tid] == 1){
			int val=0;
			int i =0;
			int count=0; 	
			int num = tid;
			//uint64_t hash[2];
			do{	
				count++;
				num /= 10;
			} while(num != 0);
			num = tid;
			char str[10];
			do{
				val=num%10 + 48;
				num/=10;
				str[count-i-1] = val;
				i++;
			}while(num !=0);	
			str[i] = 48+'\0';
			uint64_t len1 = (uint64_t) count;
			size_t len = (size_t) len1;
			MurmurHash3_x64_128(tid, str, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));
		}
	}
}

__global__ void set_bloom(bool *bit, bool *mask, uint64_t *hash_value, int m, int h, int n)
{
	int val = blockIdx.x;
	int hash = threadIdx.x;
	uint64_t filter_size = (uint64_t) m;
	uint8_t hash_no = (uint8_t) hash;
	if (val < n){
		if (mask[val] == true){
			if (hash < h){
				bit[NthHash(hash_no,*(hash_value+2*val*sizeof(uint64_t)),*(hash_value+(2*val+1)*sizeof(uint64_t)), filter_size)] = true;
			}
		}
	}
}

__global__ void check_bloom(int *found, bool *bit, bool *mask, uint64_t *hash_value, int m, int h, int n)
{
	//volatile __shared__ bool someoneFoundIt;
	//if (threadIdx.x == 0)  someoneFoundIt = found;
    	//__syncthreads();

	int val = blockIdx.x;
	int hash = threadIdx.x;
	uint64_t filter_size = (uint64_t) m;
	uint8_t hash_no = (uint8_t) hash;
	
	if (val < n && *found == 0){
		if (mask[val] == 1){
			 if (hash < h){
				if (bit[NthHash(hash_no,*(hash_value+2*val*sizeof(uint64_t)),*(hash_value+(2*val+1)*sizeof(uint64_t)), filter_size)] == 0){ 		
					atomicAdd(found, 1); //printf("here");
				}
			}
		}
	}
}

/*__global__ void get_neighbours(int src, int *no, int *neigh, uint64_t *hash_value, bool *bit, int m, int h, int n)
{	
	int num = 0;
	int next = Parent(src+n-1);
	int cur = src+n-1;
	bool dir = 1;
	*no = 0;

	get_neighs(no, src, next, cur, dir, bit, neigh, hash_value, n, num, m, h);

	printf("no value is %d\n",*no);

	while (next >= 0 && next < ((n << 1) - 1)) {
		printf("inside loop \n");
		if (next >= n - 1) { 
			*(neigh+num*sizeof(int)) = next - n + 1;
			printf("neighbour found (%d) and num is %d\n",next - n + 1,num);
			num++;
			atomicAdd(no,1);
			if (num >= 2 && *(neigh+(num-2)*sizeof(int)) == next-n+1){
				*(neigh+(num-1)*sizeof(int)) = NULL;
				atomicSub(no,1);
				num--;
				printf("the value of num is %d\n",num);
		 		break;
			}
			else{
				int next = Parent(src+n-1);
				int cur = src+n-1;
				bool dir = 1;	
			}
		}
		if (dir && CheckBloom((((long)next * n + src) << 1),hash_value,bit,h,m) == 1) { 
			cur = next;
			next = Parent(next);
		}
		if (dir && CheckBloom(((((long)next * n + src) << 1) + 1),hash_value,bit,h,m) == 1) { 
			int val = next;
			next = Sibling(cur);
			cur = val;
			dir = !dir;
		}
		if (!dir && CheckBloom((((long)next * n + src) << 1),hash_value,bit,h,m) == 1) { 
			cur = next;
			next = LeftChild(next);
		}
		if (!dir && CheckBloom(((((long)next * n + src) << 1) + 1),hash_value,bit,h,m) == 1) {
			cur = next;
			next = RightChild(next);
		}
	}
}*/

__global__ void get_neighbours(int u,int *no,bool *neighs, uint64_t *hash_value, bool *bit, int m, int h, int n, long int ful_vertices)
{
	int tid = threadIdx.x;
	int v=0;

	if (tid < n){
		
		if (tid < u) v = tid;
		else v = tid+1;

		u = u + n - 1;
		v = v + n - 1;
		int src = u;
		int dest = v;

		if (!((u < ful_vertices && v < ful_vertices) || (u >= ful_vertices && v >= ful_vertices))) {
			if (u > v){
	 			int cur = Parent(u);
				if (!(CheckBloom(((cur*n+u-n+1) << 1),hash_value,bit,h,m))) return;
				//if (u[tid] == LeftChild(cur))
				//	mask[(cur*n+v-n+1) << 1] = 1;
				//else
				//	mask[((cur*n+v-n+1) << 1) + 1] = 1;
				u = cur;
			}	
		 	else{
	 			int cur = Parent(v);
				//mask[(cur*n+v-n+1) << 1] = 1;
				if (v == LeftChild(cur))
				{
					if (!(CheckBloom(((cur*n+u-n+1) << 1),hash_value,bit,h,m))) 
						return;
				}
				else
				{
					if (!(CheckBloom((((cur*n+u-n+1) << 1) + 1),hash_value,bit,h,m))) 
						return;
				}
				v = cur;
			}
		}
	
		__syncthreads();

		int lca = calculate_lca(u, v);

		if (check_traversal_up(u, lca, src, dest, n, hash_value, bit, h, m)) 
		{
			if(check_traversal_down(v, lca, dest, src, n, hash_value, bit, h, m))
			{
				atomicAdd(no,1);
				//printf("here and v is %d\n",dest-n+1);
				*(neighs+(dest-n+1)*sizeof(bool)) = 1;
			}
		}
	}
}

__global__ void print_hash(uint64_t *hash_value, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 4*n*(n-1)){
		//printf("%u\n",*(hash_value+tid*sizeof(uint8_t)));
		printf("%" PRIu64 " %d\n",*(hash_value+tid*sizeof(uint64_t)),tid);
	}	
}

__device__ int Parent(int node)
{
	return (((node + 1) >> 1) - 1); 
}

__device__ int LeftChild(int node)
{
	return (((node + 1) << 1) - 1); 
}

__device__ int RightChild(int node)
{
	return ((node + 1) << 1); 
}

__device__ int Sibling(int node)
{
	return (((node + 1) ^ 1) - 1); 
}

__device__ int calculate_lca(int u, int v)
{	
	int val1 = 0;
	int val2 = 0;	
	int i = 1;

	do{
		float pow_val = 1 << i;
		val1 = floor((u+1)/pow_val);
		val2 = floor((v+1)/pow_val);
		i++;
	} while(val1 != val2);
	return (val1 - 1);
}

__device__ void traversal(int prev, int lca, int src, int dest, bool *mask, int n)
{
	int cur = Parent(prev);
	while (cur != lca){
		mask[(cur*n+src-n+1) << 1] = 1;
		if (prev == LeftChild(cur))
			mask[(cur*n+dest-n+1) << 1] = 1;
		else
			mask[((cur*n+dest-n+1) << 1) + 1] = 1;
		prev = cur;
		cur = Parent(cur);
	}
	mask[((cur*n+src-n+1) << 1) + 1] = 1;	
}

__device__ bool check_traversal_up(int prev, int lca, int src, int dest, int n, uint64_t *hash_value, bool *bit, int h, int m)  
{
	int cur = Parent(prev);
	while (cur != lca){
		if (!(CheckBloom(((cur*n+src-n+1) << 1),hash_value,bit,h,m)))  return false;
		/*if (prev == LeftChild(cur))
		{
			if (!(CheckBloom(((cur*n+dest-n+1) << 1),hash_value,bit,h,m))) 
				return false;
		}
		else
		{
			if (!(CheckBloom((((cur*n+dest-n+1) << 1) + 1),hash_value,bit,h,m))) 
				return false;
		}*/
		prev = cur;
		cur = Parent(cur);
	}
	if (!(CheckBloom((((cur*n+src-n+1) << 1) + 1),hash_value,bit,h,m)))  return false;
	return true;	
}

__device__ bool check_traversal_down(int prev, int lca, int src, int dest, int n, uint64_t *hash_value, bool *bit, int h, int m) 
{
	int cur = Parent(prev);
	while (cur != lca){
		//if (!(CheckBloom(((cur*n+src-n+1) << 1),hash_value,bit,h,m))) return false;
		if (prev == LeftChild(cur))
		{
			if (!(CheckBloom(((cur*n+dest-n+1) << 1),hash_value,bit,h,m)))  
				return false;
		}
		else
		{
			if (!(CheckBloom((((cur*n+dest-n+1) << 1) + 1),hash_value,bit,h,m))) 
				return false;
		}
		prev = cur;
		cur = Parent(cur);
		//printf("cur and lca is %d and %d for src %d\n",cur,lca,src);
	}
	//if (CheckBloom((((cur*n+src-n+1) << 1) + 1),hash_value,bit,h,m) == 0) return false;
	return true;	
}

/*__device__ void get_neighs(int *no, int src, int next, int cur, bool dir,bool *bit,int *neigh,uint64_t *hash_value,int n,int num,int m,int h)
{
	if (next < 0 || next >= ((n << 1) - 1)) return;
	if (next >= n - 1) { 
		*(neigh+num*sizeof(int)) = next - n + 1;
		num++;
		//printf("%d\n",next-n+1);
		atomicAdd(no,1);
		return;
	}
	if (dir && CheckBloom((int)((next * n + src) << 1),hash_value,bit,h,m) == 1) { 
		get_neighs(no, src, Parent(next), next, dir, bit, neigh, hash_value, n, num, m, h);
	}
	if (dir && CheckBloom((int)(((next * n + src) << 1) + 1),hash_value,bit,h,m) == 1) {
		get_neighs(no, src, Sibling(cur), next, !dir, bit, neigh, hash_value, n, num, m, h);
	}
	if (!dir && CheckBloom((int)((next * n + src) << 1),hash_value,bit,h,m) == 1) { 
		get_neighs(no, src, LeftChild(next), next, dir, bit, neigh, hash_value, n, num, m, h);
	}
	if (!dir && CheckBloom((int)(((next * n + src) << 1) + 1),hash_value,bit,h,m) == 1) { 
		get_neighs(no, src, RightChild(next), next, dir, bit, neigh, hash_value, n, num, m, h);
	}
}*/


__device__ bool CheckBloom(int tid, uint64_t *hash_value, bool *bit, int h, int m)
{
	//printf("%d ",tid);
	//int val=0;
	int i =0;
	int count=0; 	
	int num = tid;
	char str[10];
	while(num != 0){	
		count++;
		num /= 10;
	} 
	num = tid;
	while(num !=0){	
		str[count-i-1] = num%10 + '0';
		num/=10;
		i++;
	}	
	str[i] = 48+'\0';
	//printf("h %c\n",str[i]);
	uint64_t len1 = (uint64_t) count;
	size_t len = (size_t) len1;
	//MurmurHash3_x64_128(tid, str, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));
	MurmurHash3_x64_128(tid, str, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));
	
	for (int i=0; i<h; i++){
		if (bit[NthHash(i,*((hash_value)+tid*2*sizeof(uint64_t)),*((hash_value)+(tid*2+1)*sizeof(uint64_t)),m)] == 0){
			//printf("false %d\n",tid);
			return false;
		}
	}
	//printf("check true is %d\n",tid);
	return true;
}

void InsertEdge(int num_vertices, int num_edges, int num_hashes, int num_bits, int *h_u, int *h_v, bool *h_bits)
{

	size_t size = num_edges * sizeof(int);
	int num_vals = 2*num_vertices*(num_vertices-1);

	int *d_u = NULL, *d_v = NULL; 
        hipMalloc((void **)&d_u, size);
	hipMalloc((void **)&d_v, size);
	hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
	hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);

	size_t size_mask = num_vals * sizeof(bool);
	bool *d_mask = NULL;
        hipMalloc((void **)&d_mask, size_mask);
	dim3 tpb(num_vertices,(2*(num_vertices-1)),1);
	dim3 bpg(1,1,1);
	//dim3 tpb(32,32,1);
	//dim3 bpg(num_vertices/32,(num_vertices-1)/32,2);
	init_mask<<<bpg,tpb>>>(d_mask,num_vertices);

	//int tpb1 = 1024;
        //int bpg1 = num_edges/1024;
	int tpb1 = num_edges;
        int bpg1 = 1;
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;
	get_mask<<<bpg1,tpb1>>>(d_u,d_v,d_mask,num_vertices,num_edges,ful_vertices);	
	hipFree(d_u);
	hipFree(d_v);

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);
	get_hash<<<bpg,tpb>>>(d_mask,d_hash_value,num_vertices);

	size_t size_bits = num_bits * sizeof(bool);
	bool *d_bits = NULL;
        hipMalloc((void **)&d_bits, size_bits);
	//cudaMemset(d_bits, 0, size_bits);
	init_bits<<<1,num_bits>>>(d_bits,num_bits);
	set_bloom<<<num_vals,num_hashes>>>(d_bits,d_mask,d_hash_value,num_bits,num_hashes,num_vals);
	
	hipDeviceSynchronize();	
	hipMemcpy(h_bits, d_bits, size_bits, hipMemcpyDeviceToHost);

	//dim3 tpb2(num_vertices,(4*(num_vertices-1)),1);
	//dim3 bpg2(1,1,1);
	//print_hash<<<bpg2,tpb2>>>(d_hash_value,num_vertices);
	
	//for (int i =0; i<num_bits; i++){
	//	if (h_bits[i] == true) printf("%d, %d\n",i, h_bits[i]);
	//}

	hipFree(d_mask);
	hipFree(d_hash_value);
	hipFree(d_bits);
}

bool IsEdge(int u, int v, int num_vertices, int num_hashes, int num_bits, bool *h_bits)
{
	size_t size = sizeof(int);
	int *d_u = NULL, *d_v = NULL;
	int *h_eu = (int *)malloc(size);
	int *h_ev = (int *)malloc(size); 
        hipMalloc((void **)&d_u, size);
	hipMalloc((void **)&d_v, size);
	h_eu[0] = u;
	h_ev[0] = v;
	hipMemcpy(d_u,h_eu,size,hipMemcpyHostToDevice);
	hipMemcpy(d_v,h_ev,size,hipMemcpyHostToDevice);

	int num_vals = 2*num_vertices*(num_vertices-1);
	size_t size_mask = num_vals * sizeof(bool);
	bool *d_mask = NULL;
        hipMalloc((void **)&d_mask, size_mask);
	dim3 tpb(num_vertices,(2*(num_vertices-1)),1);
	dim3 bpg(1,1,1);
	init_mask<<<bpg,tpb>>>(d_mask,num_vertices);

	int tpb1 = 1;
        int bpg1 = 1;
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;
	get_mask<<<bpg1,tpb1>>>(d_u,d_v,d_mask,num_vertices,1,ful_vertices);
	
	hipFree(d_u);
	hipFree(d_v);
	free(h_eu);
	free(h_ev);

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);
	get_hash<<<bpg,tpb>>>(d_mask,d_hash_value,num_vertices);

	size_t size_bits = num_bits * sizeof(bool);
	bool *d_bits = NULL;
        hipMalloc((void **)&d_bits, size_bits);
	hipMemcpy(d_bits, h_bits, size_bits, hipMemcpyHostToDevice);

	size_t size_f = sizeof(int);
	int *d_found = NULL;
	int *h_found = (int *)malloc(size_f);
        hipMalloc((void **)&d_found, size_f);
	h_found[0] = 0;
	hipMemcpy(d_found,h_found,size,hipMemcpyHostToDevice);
	check_bloom<<<num_vals,num_hashes>>>(d_found,d_bits,d_mask,d_hash_value,num_bits,num_hashes,num_vals);

	hipFree(d_mask);
	hipFree(d_hash_value);
	hipFree(d_bits);

	hipMemcpy(h_found,d_found,size,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	if (*h_found>0) return 0;
	else return 1;

}

void Neighbours(int u, int num_vertices, int num_hashes, int num_bits, bool *h_bits)
{
	/*size_t size = sizeof(int);
	int *d_u = NULL;
	int *h_nu = (int *)malloc(size);
        cudaMalloc((void **)&d_u, size);
	h_nu[0] = u;
	cudaMemcpy(d_u,h_nu,size,cudaMemcpyHostToDevice);*/

	//cudaError_t err = cudaSuccess;

	int num_vals = 2*num_vertices*(num_vertices-1);

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	hipMalloc((void **)&d_hash_value, size_hash);

	size_t size_bits = num_bits * sizeof(bool);
	bool *d_bits = NULL;
        hipMalloc((void **)&d_bits, size_bits);
	hipMemcpy(d_bits, h_bits, size_bits, hipMemcpyHostToDevice);

	bool *d_neighs = NULL;
	size_t size_neighs = (num_vertices)*sizeof(bool);
	bool *h_neighs = (bool *)malloc(size_neighs);
	hipMalloc((void **)&d_neighs, size_neighs);

	int *d_no = NULL;
	size_t size_no = sizeof(int);
	int *h_no = (int *)malloc(size_no);
	hipMalloc((void **)&d_no, size_no);
	//get_neighbours<<<1,1>>>(u,d_no,d_neighs,d_hash_value,d_bits,num_bits,num_hashes,num_vertices);
	
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;

	init_bits<<<1,num_vertices>>>(d_neighs,num_vertices);
	get_neighbours<<<1,num_vertices-1>>>(u,d_no,d_neighs,d_hash_value,d_bits,num_bits,num_hashes,num_vertices,ful_vertices);

	hipMemcpy(h_no, d_no, size_no, hipMemcpyDeviceToHost);
	hipMemcpy(h_neighs, d_neighs, size_neighs, hipMemcpyDeviceToHost);
	
	printf("the no of neighbours are %d\n", *h_no);
	printf("The neighbours are: \n");
	for (int i=0; i<num_vertices; i++)
	{
		if (h_neighs[i] == 1)
			printf("%d\n",i);

	}
}

int main ()
{

	int num_vertices, num_edges, num_hashes, num_bits;
	scanf("%d",&num_vertices);
	scanf("%d",&num_edges);
	scanf("%d",&num_bits);
	scanf("%d",&num_hashes);

	size_t size = num_edges * sizeof(int);
	//int num_vals = 2*num_vertices*(num_vertices-1);

	int *h_u = (int *)malloc(size);
	int *h_v = (int *)malloc(size);
	
	for (int i =0; i<num_edges; i++)
	{
		scanf("%d",&h_u[i]);
		scanf("%d",&h_v[i]);
	}

	size_t size_bits = num_bits * sizeof(bool);
	bool *h_bits = (bool *)malloc(size_bits);

	InsertEdge(num_vertices, num_edges, num_hashes, num_bits, h_u, h_v, h_bits);

	bool val = IsEdge(8, 7, num_vertices, num_hashes, num_bits, h_bits);
	if (val == 0) printf("It is NOT an edge.\n");
	else printf("It is an edge.\n");

	Neighbours(8, num_vertices, num_hashes, num_bits, h_bits);

	free(h_u);
	free(h_v);
	free(h_bits);
	
	hipDeviceReset();
	return 0;
}
