#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdint.h>
#include <inttypes.h>

#define FORCE_INLINE

__device__ int Parent(int node);
__device__ int LeftChild(int node);
__device__ int RightChild(int node);
__device__ int Sibling(int node);

__device__ static inline FORCE_INLINE uint64_t rotl64 ( uint64_t x, int8_t r )
{
  return (x << r) | (x >> (64 - r));
}

#define ROTL64(x,y)	rotl64(x,y)
#define BIG_CONSTANT(x) (x##LLU)

#define getblock(p, i) (p[i])

__device__ static inline FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

__device__ void MurmurHash3_x64_128 ( const char key[], const int len,
                           const uint32_t seed, void * out1 , void * out2)
{
  //const uint8_t * data = (const uint8_t*)key;

  uint8_t data[10];
  for (int i=0; i<len; i++)
    data[i] = (uint8_t) key[i];

  const int nblocks = len / 16;
  int i;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //----------
  // body

  //const uint64_t * blocks = (const uint64_t *)(data);

  uint64_t blocks[10];
  for (int i=0; i<len; i++)
    blocks[i] = (uint64_t) data[i];

  for(i = 0; i < nblocks; i++)
  {
    uint64_t k1 = blocks[i*2+0];
    uint64_t k2 = blocks[i*2+1];

    k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;

    h1 = ROTL64(h1,27); h1 += h2; h1 = h1*5+0x52dce729;

    k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    h2 = ROTL64(h2,31); h2 += h1; h2 = h2*5+0x38495ab5;
  }

  //----------
  // tail

  //const uint8_t * tail = (const uint8_t*)(data + nblocks*16);

  uint8_t tail[10];
  for (int i=0; i<len-nblocks*16; i++)
    tail[i] = (uint64_t) data[i+nblocks*16];

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15)
  {
  case 15: k2 ^= (uint64_t)(tail[14]) << 48;
  case 14: k2 ^= (uint64_t)(tail[13]) << 40;
  case 13: k2 ^= (uint64_t)(tail[12]) << 32;
  case 12: k2 ^= (uint64_t)(tail[11]) << 24;
  case 11: k2 ^= (uint64_t)(tail[10]) << 16;
  case 10: k2 ^= (uint64_t)(tail[ 9]) << 8;
  case  9: k2 ^= (uint64_t)(tail[ 8]) << 0;
           k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

  case  8: k1 ^= (uint64_t)(tail[ 7]) << 56;
  case  7: k1 ^= (uint64_t)(tail[ 6]) << 48;
  case  6: k1 ^= (uint64_t)(tail[ 5]) << 40;
  case  5: k1 ^= (uint64_t)(tail[ 4]) << 32;
  case  4: k1 ^= (uint64_t)(tail[ 3]) << 24;
  case  3: k1 ^= (uint64_t)(tail[ 2]) << 16;
  case  2: k1 ^= (uint64_t)(tail[ 1]) << 8;
  case  1: k1 ^= (uint64_t)(tail[ 0]) << 0;
           k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)out1)[0] = h1;
  ((uint64_t*)out2)[0] = h2;

}

__device__ inline uint64_t NthHash(uint8_t n, uint64_t hashA, uint64_t hashB, uint64_t filter_size) {
	//printf("%u and %u\n",hashA , hashB);
	return (hashA + n * hashB) % filter_size;
}

__global__ void init_mask(bool *mask, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;
	if (tid < 2*n*(n-1))
		mask[tid] = 0;
}

__global__ void check_mask(bool *mask, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;
	if (tid < 2*n*(n-1))
		mask[tid] = 0;
}

__global__ void adjust_level(int *u, int *v, bool *mask, int n, int e, long int ful_vertices)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid < e){
		u[tid] = u[tid] + n - 1;
		v[tid] = v[tid] + n - 1;
		__syncthreads();
		if (!((u[tid] < ful_vertices && v[tid] < ful_vertices) || (u[tid] >= ful_vertices && v[tid] >= ful_vertices))) {
			if (u[tid] > v[tid]){
				int src = u[tid];
				int dest = v[tid];
	 			int cur = Parent(u[tid]);
				mask[(cur*n+src-n+1) << 1] = 1;
				if (src == LeftChild(cur))
					mask[(cur*n+dest-n+1) << 1] = 1;
				else
					mask[((cur*n+dest-n+1) << 1) + 1] = 1;
				u[tid] = cur;
			}	
		 	else{
				int src = v[tid];
				int dest = u[tid];
	 			int cur = Parent(v[tid]);
				mask[(cur*n+src-n+1) << 1] = 1;
				if (src == LeftChild(cur))
					mask[(cur*n+dest-n+1) << 1] = 1;
				else
					mask[((cur*n+dest-n+1) << 1) + 1] = 1;
				v[tid] = cur;
			}
		}
	}
}

__global__ void calculate_lca(int *u, int *v, int *lca, int e)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	int val1 = 0;
	int val2 = 0;	

	if (tid < e){
		int i = 1;
		do{
			float pow_val = 1 << i;
			val1 = floor((u[tid]+1)/pow_val);
			val2 = floor((v[tid]+1)/pow_val);
			i++;
		} while(val1 != val2);
		lca[tid] = val1 - 1;
	}
}

__global__ void traversal(int *u, int *lca, int *source, int *destination, bool *mask, int n, int e)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;		

	if (tid < e){
		int prev = u[tid];
		int cur = Parent(u[tid]);
		int src = source[tid];
		int dest = destination[tid];
		while (cur != lca[tid]){
			mask[(cur*n+src) << 1] = 1;
			if (prev == LeftChild(cur))
				mask[(cur*n+dest) << 1] = 1;
			else
				mask[((cur*n+dest) << 1) + 1] = 1;
			prev = cur;
			cur = Parent(cur);
		}
		mask[((cur*n+src) << 1) + 1] = 1;
		mask[((cur*n+dest) << 1) + 1] = 1;
	}	
}

__global__ void to_str(char *str, bool *mask, int n, int count)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid<n){
		int num=tid;
		int val=0;
		int i =0;
		int len=0;
		do{
			len++;
			num/=10;
		}while(num !=0);
		num = tid;
		do{
			val=num%10 + 48;
			num/=10;
			char *op = (char *) (str+tid*sizeof(char)*(count+1)+len-i-1);
			*op = val;
			i++;
		}while(num !=0);
		char *op = (char *) (str+tid*sizeof(char)*(count+1)+len);
		*op = 48+'\0';
	}
}

__global__ void get_hash(bool *mask, uint64_t *hash_value, char *array, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 2*n*(n-1)){
		if (mask[tid] == 1){
			int val=0;
			int i =0;
			int count=0; 	
			int num = tid;
			//uint64_t hash[2];
			do{	
				count++;
				num /= 10;
			} while(num != 0);
			num = tid;
			char str[10];
			do{
				val=num%10 + 48;
				num/=10;
				str[i] = val;
				i++;
			}while(num !=0);
			str[i] = 48+'\0';
			uint64_t len1 = (uint64_t) count;
			size_t len = (size_t) len1;
			MurmurHash3_x64_128(str, len, 0, (hash_value)+tid*2*sizeof(uint64_t), (hash_value)+(tid*2+1)*sizeof(uint64_t));
		}
	}
}

__global__ void set_bloom(bool *bit, bool *mask, uint64_t *hash_value, int m, int h, int n)
{
	int val = blockIdx.x;
	int hash = threadIdx.x;
	uint64_t filter_size = (uint64_t) m;
	uint8_t hash_no = (uint8_t) hash;
	
	if (val < n){
		if (mask[val] == 1){
			if (hash < h){
				bit[NthHash(hash_no,*(hash_value+2*val*sizeof(uint64_t)),*(hash_value+(2*val+1)*sizeof(uint64_t)), filter_size)] = 1;
			}
		}
	}
}	

__global__ void print_hash(uint64_t *hash_value, int n)
{
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*gridDim.x+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
	int tid = blockNum*(blockDim.x*blockDim.y*blockDim.z)+threadNum;

	if (tid < 4*n*(n-1)){
		//printf("%u\n",*(hash_value+tid*sizeof(uint8_t)));
		printf("%" PRIu64 " %d\n",*(hash_value+tid*sizeof(uint64_t)),tid);
	}	
}

__device__ int Parent(int node)
{
	return (((node + 1) >> 1) - 1); 
}

__device__ int LeftChild(int node)
{
	return (((node + 1) << 1) - 1); 
}

__device__ int RightChild(int node)
{
	return ((node + 1) << 1); 
}

__device__ int Sibling(int node)
{
	return (((node + 1) ^ 1) - 1); 
}

int main ()
{
	hipError_t err = hipSuccess;

	int num_vertices, num_edges, num_hashes, num_bits;
	scanf("%d",&num_vertices);
	scanf("%d",&num_edges);
	scanf("%d",&num_bits);
	scanf("%d",&num_hashes);

	size_t size = num_edges * sizeof(int);
	int num_vals = 2*num_vertices*(num_vertices-1);

	int *h_u = (int *)malloc(size);
	int *h_v = (int *)malloc(size);

	if (h_u == NULL || h_u == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors u and v!\n");
            exit(EXIT_FAILURE);
        }
	
	for (int i =0; i<num_edges; i++)
	{
		scanf("%d",&h_u[i]);
		scanf("%d",&h_v[i]);
	}

	int *d_u = NULL, *d_v = NULL, *d_src = NULL, *d_dest = NULL;
        err= hipMalloc((void **)&d_u, size);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector u (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err= hipMalloc((void **)&d_v, size);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector v (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err= hipMalloc((void **)&d_src, size);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector src (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err= hipMalloc((void **)&d_dest, size);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector dest (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	err= hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector u from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector v from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipMemcpy(d_src, h_u, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector src from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipMemcpy(d_dest, h_v, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector dest from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	size_t size_mask = num_vals * sizeof(bool);
	//bool *h_mask = (bool *)malloc(size_mask);
	bool *d_mask = NULL;
        err= hipMalloc((void **)&d_mask, size_mask);

	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector mask (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	dim3 tpb(num_vertices,(2*(num_vertices-1)),1);
	dim3 bpg(1,1,1);

	//dim3 tpb(32,32,1);
	//dim3 bpg(num_vertices/32,(num_vertices-1)/32,2);

	init_mask<<<bpg,tpb>>>(d_mask,num_vertices);

	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch init_mask kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	//int tpb1 = 1024;
        //int bpg1 = num_edges/1024;
	int tpb1 = num_edges;
        int bpg1 = 1;
	int num_ful_levels = floor( log2((double) (2*num_vertices - 1)));
	long int ful_vertices = pow((int) 2,(int) num_ful_levels) - 1;

	adjust_level<<<bpg1,tpb1>>>(d_u,d_v,d_mask,num_vertices,num_edges,ful_vertices);

	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch adjust_level kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }	

	int *d_lca = NULL;
        err= hipMalloc((void **)&d_lca, size);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	calculate_lca<<<bpg1,tpb1>>>(d_u,d_v,d_lca,num_edges);
	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch calculate_lca kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	traversal<<<bpg1,tpb1>>>(d_u,d_lca,d_src,d_dest,d_mask,num_vertices,num_edges);
	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch traversal kernel 1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	traversal<<<bpg1,tpb1>>>(d_v,d_lca,d_dest,d_src,d_mask,num_vertices,num_edges);
	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch traversal kernel 2 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	
	err= hipFree(d_lca);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipFree(d_u);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipFree(d_v);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipFree(d_src);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	err= hipFree(d_dest);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector lca (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }	

	uint64_t *d_hash_value = NULL;
	size_t size_hash = 2*num_vals*sizeof(uint64_t);
	err= hipMalloc((void **)&d_hash_value, size_hash);
	//uint64_t *h_hash_value = (uint64_t *)malloc(size_hash);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector hash_value (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	int num = num_vals;
	int count = 0;
	do{
		count++;
		num /= 10;
	} while(num != 0);
	
	char *d_str = NULL;
	size_t size_str = (count+1)*num_vals*sizeof(char);
	err= hipMalloc((void **)&d_str, size_str);
	//char *h_str = (char *)malloc(size_str);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector str (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }	

	get_hash<<<bpg,tpb>>>(d_mask,d_hash_value,d_str,num_vertices);
	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch get_hash kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	size_t size_bits = num_bits * sizeof(bool);
	bool *h_bits = (bool *)malloc(size_bits);

	if (h_bits == NULL)
        {
            fprintf(stderr, "Failed to allocate host vector bits!\n");
            exit(EXIT_FAILURE);
        }

	bool *d_bits = NULL;
        err= hipMalloc((void **)&d_bits, size_bits);
	if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector bits (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

	set_bloom<<<num_vals,num_hashes>>>(d_bits,d_mask,d_hash_value,num_bits,num_hashes,num_vals);
	err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch set_bloom kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	
	dim3 tpb2(num_vertices,(4*(num_vertices-1)),1);
	dim3 bpg2(1,1,1);
	print_hash<<<bpg2,tpb2>>>(d_hash_value,num_vertices);
	
	hipDeviceSynchronize();	
	hipMemcpy(h_bits, d_bits, size_bits, hipMemcpyDeviceToHost);

	hipFree(d_mask);
	hipFree(d_str);
	hipFree(d_hash_value);
	hipFree(d_bits);

	free(h_u);
	free(h_v);
	free(h_bits);
	
	hipDeviceReset();
	return 0;
}